/*
* Copyright 2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/
#include <claraparabricks/genomeworks/cudaungappedextender/cudaungappedextender.hpp>
#include <claraparabricks/genomeworks/io/fasta_parser.hpp>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string>
#include <vector>

using namespace claraparabricks::genomeworks;
using namespace claraparabricks::genomeworks::cudaungappedextender;

int main(int argc, char* argv[])
{
    const int32_t input_xdrop = 910;
    const bool input_no_entropy = false;
    const int32_t score_threshold = 3000;

    // Define an alphabet for the sequences to be processed
    Alphabet alphabet  = make_alphabet("acgt");

    // Fasta query and target files
    std::string target_file_path = "../data/example.fa";
    std::unique_ptr<io::FastaParser> fasta_parser_target =
        io::create_kseq_fasta_parser(alphabet, target_file_path, 0, false);
    // Assumes that only one sequence is present per file
    SequenceVector target_sequences = fasta_parser_target->get_sequence_by_id(0);

    std::string query_file_path = "../data/example.fa";
    std::unique_ptr<io::FastaParser> fasta_parser_query =
        io::create_kseq_fasta_parser(alphabet, query_file_path, 0, false);
    // Assumes that only one sequence is present per file
    SequenceVector query_sequences = fasta_parser_query->get_sequence_by_id(0);

    // CSV SeedPairs file - Each row -> query_position_in_read_,
    // target_position_in_read_
    std::string seed_pairs_file_path = "../data/example_seed_pairs.csv";

    std::vector<SeedPair> h_seed_pairs;
    // Following function loops through all seed_pairs in the SeedPairs csv and returns
    // results in
    // the passed vector
    parse_SeedPairs(seed_pairs_file_path, h_seed_pairs);

    // Following sections TBD based on encoding
    ScoreMatrix score_matrix(a);
    score_matrix('a','a') = score;
    score_matrix('a','c') = score;
    score_matrix('a','g') = score;
    score_matrix('a','t') = score;
    score_matrix('c','c') = score;
    score_matrix('c','g') = score;
    score_matrix('c','t') = score;
    score_matrix('g','g') = score;
    score_matrix('g','t') = score;
    score_matrix('t','t') = score;

    // Create a stream for async use
    CudaStream stream0 = make_cuda_stream();
    // Create an ungapped extender object
    std::unique_ptr<UngappedExtender> ungapped_extender =
        std::make_unique<UngappedExtender>(0, score_matrix, input_xdrop,
                                           input_no_entropy, stream0.get());
    // Launch the ungapped extender host function
    int32_t query_idx  = 0;
    int32_t target_idx = 0;
    ungapped_extender->extend_async(query_sequences, query_idx, target_sequences, target_idx, score_threshold, h_seed_pairs);

    // Wait for ungapped extender to finish
    ungapped_extender->sync();

    // Get results
    const std::vector<ScoredSegmentPair>& segments =
        ungapped_extender->get_scored_segment_pairs();
    int32_t i = 0;
    for (const auto& segment : segments)
    {
        std::cout << "Segment: " << i << "Length: " << segment.length
                  << "Score: " << segment.score << std::endl;
        std::cout << "Position in query: "
                  << segment.seed_pair.query_position_in_read << std::endl;
        std::cout << "Position in target: "
                  << segment.seed_pair.target_position_in_read << std::endl;
        i++;
    }

    return 0;
}
